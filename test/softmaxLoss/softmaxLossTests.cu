#include <complex.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include <iostream>
#include <random>

#include "../../src/cudaHelpers.cuh"
#include "../../src/softmaxLoss.cuh"

#define MINIBATCHSIZE 100
#define CLASSES 10

using namespace std;

// function prototypes
void warmUpGPU();
void compareMatrices(float *C_GPU, float *C_CPU, unsigned int NUMELEM);
void printMatrix(float *matrix, int width, int height);
void outputSumElems(float *C, unsigned int NUMELEM);

int main(int argc, char *argv[]) {
    warmUpGPU();

    // seed random number generator with constant seed
    srand(123);

    // Intermediate Scores f(x). The linear classifier's predicted scores f(x)=W*x+b
    float *dev_f1;
    gpuErrchk(hipMalloc((float **)&dev_f1, sizeof(float) * CLASSES));

    // The expected classes of the minibatch, used to train the model
    float *dev_y;
    gpuErrchk(hipMalloc((float **)&dev_y, sizeof(float) * MINIBATCHSIZE));

    // Softmax loss
    float *dev_softmax_loss;
    gpuErrchk(hipMalloc((float **)&dev_softmax_loss, sizeof(float)));

    // Softmax dL/df. How much the loss changes with respect to each class score from the last layer
    float *dev_dLdf;
    gpuErrchk(hipMalloc((float **)&dev_dLdf, sizeof(float) * CLASSES));

    softmaxLoss_t *softmaxInputs;
    softmaxInputs->loss = dev_softmax_loss;
    softmaxInputs->dLdf = dev_dLdf;
    softmaxInputs->f = dev_f1;
    softmaxInputs->numClasses = CLASSES;
    softmaxInputs->batchSize = MINIBATCHSIZE;

    // ****** Initialize Model Parameters *********

    // Set scores to small values, gaussian distribution, 0 mean
    float scale = 1.0;
    int scoresSize = sizeof(float) * CLASSES * MINIBATCHSIZE;
    float *host_scores = (float *)malloc(scoresSize);
    std::default_random_engine generator;
    std::normal_distribution<float> distribution(0.0, scale);

    for (int i = 0; i < CLASSES * MINIBATCHSIZE; i++) {
        host_scores[i] = distribution(generator);
    }
    // printMatrix(host_scores, MINIBATCHSIZE, CLASSES);

    // Copy scores to device
    gpuErrchk(hipMemcpy(dev_f1, host_scores, scoresSize, hipMemcpyHostToDevice));

    double tstart = omp_get_wtime();

    // execute kernel
    softmaxLoss(softmaxInputs);

    // end execute kernel

    double tend = omp_get_wtime();

    // Copy Loss off GPU
    float *host_loss;
    host_loss = (float *)malloc(sizeof(float));
    gpuErrchk(hipMemcpy(host_loss, dev_softmax_loss, sizeof(float), hipMemcpyDeviceToHost));
    printf("Softmax Loss: %f\n", *host_loss);

    // Copy gradient off GPU
    float *host_gradient;
    host_gradient = (float *)malloc(sizeof(float) * CLASSES);
    gpuErrchk(hipMemcpy(host_gradient, dev_dLdf, sizeof(float) * CLASSES, hipMemcpyDeviceToHost));
    printf("dL/df: \n");
    printMatrix(host_gradient, 1, CLASSES);

    printf("\nTotal time GPU (s): %f", tend - tstart);

    printf("\n");

    // free memory

    return 0;
}

void warmUpGPU() {
    printf("Warming up GPU for time trialing...");
    hipDeviceSynchronize();
    return;
}

void printMatrix(float *matrix, int width, int height) {
    int i, j;
    int cnt = 0;
    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            printf("%.2f, ", matrix[cnt]);
            cnt++;
        }
        printf("\n");
    }
}
