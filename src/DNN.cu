#include "hip/hip_runtime.h"
// To run this program, pass in the file path to the dataset and the neural network will train on
// the dataset, and then the accuracy will be evaluated.

#include <complex.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include <random>
#include <vector>

#include "ReLULayer.cuh"
#include "affineLayer.cuh"
#include "cudaHelpers.cuh"
#include "dataset.h"
#include "params.h"
#include "softmaxLoss.cuh"

// Number of classes to predict for on output layer
#define CLASSES 10
// Size of the NN input layer (The size of the flattened image)
#define INPUTSIZE 3072
#define TRAINSIZE 10000
#define MINIBATCHSIZE 1000
#define NUMEPOCHS 100

// Hyper parameters
#define LEARNINGRATE 0.001
#define ALPHA 0.00001
#define MOMENTUMDECAY 0.75

/* Function Prototypes */
void forward(affineInputs_t *aff1Inputs);

using namespace std;

int main(int argc, char *argv[]) {
    char filename[256];
    char *endptr;
    if (argc < 2) {
        fprintf(stderr, "Too few command line arguments. <pathToDataset> is required\n");
        return EXIT_FAILURE;
    }
    strcpy(filename, argv[1]);

    data_t *dataset = importDataset(filename, 0.6);
    // Should have a training dataset and a validation dataset. x are our inputs, y are the expected
    // outputs for each given input

    // ********* Construct the network *************
    // The network is essentially constructed from memory allocated to store all the data as it
    // propagates through the different layers, and the kernels that implement the different
    // layers, affine, ReLu, softmax, convolutional

    // Allocate memory for all intermediate steps on the GPU. This includes caching inputs to
    // each layer, outputs, and gradients used for backpropagation Input layer
    float *dev_x1;
    gpuErrchk(hipMalloc((float **)&dev_x1, sizeof(float) * MINIBATCHSIZE * INPUTSIZE));

    // W1. The weight matrix we are trying to find
    float *dev_W1;
    gpuErrchk(hipMalloc((float **)&dev_W1, sizeof(float) * CLASSES * INPUTSIZE));

    // b1. The biases for each output of the linear classifier. The +b term
    float *dev_b1;
    gpuErrchk(hipMalloc((float **)&dev_b1, sizeof(float) * CLASSES));

    // Intermediate Scores f(x). The linear classifier's predicted scores f(x)=W*x+b
    float *dev_f1;
    gpuErrchk(hipMalloc((float **)&dev_f1, sizeof(float) * CLASSES));

    affineInputs_t *aff1Inputs;
    aff1Inputs->W = dev_W1;
    aff1Inputs->x = dev_x1;
    aff1Inputs->b = dev_b1;
    aff1Inputs->f = dev_f1;
    aff1Inputs->batchSize = MINIBATCHSIZE;
    aff1Inputs->dataSize = INPUTSIZE;
    aff1Inputs->numOutputs = CLASSES;

    // dL/dW1. How much the weights effect the loss
    float *dev_dLdW1;
    gpuErrchk(hipMalloc((float **)&dev_dLdW1, sizeof(float) * CLASSES * INPUTSIZE));

    // dL/db1. How much the biases effect the loss
    float *dev_dLdb1;
    gpuErrchk(hipMalloc((float **)&dev_dLdb1, sizeof(float) * CLASSES));

    // dL/dx1. How much the inputs effect the loss
    float *dev_dLdx1;
    gpuErrchk(hipMalloc((float **)&dev_dLdx1, sizeof(float) * MINIBATCHSIZE * INPUTSIZE));

    AffineGradients *aff1Grads;
    aff1Grads->dLdB = dev_dLdb1;
    aff1Grads->dLdW = dev_dLdW1;
    aff1Grads->dLdx = dev_dLdx1;

    // The expected classes of the minibatch, used to train the model
    float *dev_y;
    gpuErrchk(hipMalloc((float **)&dev_y, sizeof(float) * MINIBATCHSIZE));

    // Softmax loss
    float *dev_softmax_loss;
    gpuErrchk(hipMalloc((float **)&dev_softmax_loss, sizeof(float)));

    // Softmax dL/df. How much the loss changes with respect to each class score from the last layer
    float *dev_dLdf;
    gpuErrchk(hipMalloc((float **)&dev_dLdf, sizeof(float) * CLASSES));

    softmaxLoss_t *softmaxInputs;
    softmaxInputs->loss = dev_softmax_loss;
    softmaxInputs->dLdf = dev_dLdf;
    softmaxInputs->f = dev_f1;
    softmaxInputs->numClasses = CLASSES;
    softmaxInputs->batchSize = MINIBATCHSIZE;

    // ****** Initialize Model Parameters *********

    // W1 needs to be set to small values, gaussian distribution, 0 mean
    float weightScale = 0.001;
    int W1Size = sizeof(float) * CLASSES * INPUTSIZE;
    float *host_W1 = (float *)malloc(W1Size);
    std::default_random_engine generator;
    std::normal_distribution<float> distribution(0.0, weightScale);

    for (int i = 0; i < CLASSES * INPUTSIZE; i++) {
        host_W1[i] = distribution(generator);
    }
    // Copy W1 to device
    gpuErrchk(hipMemcpy(dev_W1, host_W1, W1Size, hipMemcpyHostToDevice));

    // b1 needs to be set to 0 for no offsets at first
    int b1Size = sizeof(float) * CLASSES;
    float *host_b1 = (float *)malloc(b1Size);
    for (int i = 0; i < CLASSES; i++) {
        host_b1[i] = 0.0;
    }
    // Copy b1 to device
    gpuErrchk(hipMemcpy(dev_b1, host_b1, b1Size, hipMemcpyHostToDevice));

    // ******** Start of Optimization ************

    learnParams_t *learnParameters;
    learnParameters->learningRate = LEARNINGRATE;
    learnParameters->momentumDecay = MOMENTUMDECAY;
    learnParameters->regStrength = ALPHA;

    // Train for this many epochs
    for (int epoch = 0; epoch < NUMEPOCHS; epoch++) {
        // Iterate through as many minibatches as we need to complete an entire epoch
        for (int batch = 0; batch < ceil(1.0 * dataset->yTrain->size / MINIBATCHSIZE); batch++) {
            // Sample a minibatch of samples from training data
            unsigned int minibatchSize = MINIBATCHSIZE * INPUTSIZE;
            char *minibatch = (char *)malloc(sizeof(char) * minibatchSize);

            // TODO Sample the minibatch randomly from xTrain, and don't get any repeat inputs until
            // we are onto the next epoch

            // Push minibatch to GPU. Push images and expected classes
            gpuErrchk(hipMemcpy(dev_x1, minibatch, sizeof(char) * minibatchSize,
                                 hipMemcpyHostToDevice));

            // Run forward and backward passes on minibatch of data, and update the gradient

            forward(aff1Inputs);
            // ReLU next
            // Another Affine layer

            // This layer computes the loss and the gradient of the loss with respect to the scores
            // input to this layer
            softmaxLoss(softmaxInputs);

            // At this point we will have the loss computed for every input image, and the gradient
            // of our softmax function. We now begin to backpropogate the gradients

            // Evaluate gradient for affine layer with respect to W and b f(x)=W*x+b, given the
            // upstream gradients and the last inputs
            dim3 blockDim(32, 32);
            // Number of threads is the size of the output matrix
            dim3 gridDim(ceil(1.0 * MINIBATCHSIZE / blockDim.x), ceil(1.0 * CLASSES / blockDim.y));
            affineBackward<<<gridDim, blockDim>>>(dev_dLdf, aff1Inputs, aff1Grads);

            // Using our learning rate, update our parameters based on the gradient

            // Update Affine1 layer weights
            dim3 blockDim(32, 32);
            dim3 gridDim(ceil(1.0 * MINIBATCHSIZE / blockDim.x), ceil(1.0 * CLASSES / blockDim.y));
            affineUpdate<<<gridDim, blockDim>>>(learnParameters, aff1Inputs, aff1Grads);

            // Print out the loss for debugging
            float loss;
            gpuErrchk(hipMemcpy(&loss, dev_softmax_loss, sizeof(float), hipMemcpyDeviceToHost));
            printf("\nSoftmax Loss: %f", loss);
        }
    }

    // TODO Optional, save model off so we don't have to retrain in the future

    // Evaluate accuracy of classifier on training dataset
    float trainAccuracy;

    // TODO Run all the xTrain data through the model and evaluate the accuracy
    forward(aff1Inputs);

    printf("Train Accuracy: %f\n", trainAccuracy);

    // Evaluate accuracy of classifier on validation dataset
    float valAccuracy;

    // TODO Do the same for xVal and yVal and evaluate accuracy

    printf("Validation Accuracy: %f\n", valAccuracy);

    // Cleanup, free memory etc
}

/*! \brief Compute the forward pass
 *
 *  Used during training as well as for evaluating model performance. Evaluate forward pass for
 * entire network
 *
 * \param aff1Inputs Inputs for first affine layer
 * \return void
 */
void forward(affineInputs_t *aff1Inputs) {
    // Compute f(x)=W1*x+b1 forward pass
    dim3 blockDim(32, 32);
    // Number of threads is the size of the output matrix
    dim3 gridDim(ceil(1.0 * aff1Inputs->batchSize / blockDim.x),
                 ceil(1.0 * aff1Inputs->numOutputs / blockDim.y));
    affineForward<<<gridDim, blockDim>>>(aff1Inputs);
}
