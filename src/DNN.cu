#include "hip/hip_runtime.h"
// To run this program, pass in the file path to the dataset and the neural network will train on
// the dataset, and then the accuracy will be evaluated.
#include <complex.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <random>
#include <vector>

#include "ReLULayer.cuh"
#include "affineLayer.cuh"
#include "cudaHelpers.cuh"
#include "dataset.h"
#include "params.h"
#include "softmaxLoss.cuh"

// Number of classes to predict for on output layer
#define CLASSES 10
// Size of the NN input layer (The size of the flattened image)
#define INPUTSIZE 3072
#define HIDDENLAYERSIZE 100
#define MINIBATCHSIZE 1000
#define NUMEPOCHS 40

#define TEST 1

// Hyper parameters
#define MOMENTUMDECAY 0.9

/* Function Prototypes */
void forward(affineInputs_t *aff1Inputs, reluInput_t *reluInputs, affineInputs_t *aff2Inputs);
void backward(float regStrength, float *dLdf, affineInputs_t *aff1Inputs, reluInput_t *reluInputs,
              affineInputs_t *aff2Inputs);
void update(learnParams_t *learnParameters, affineInputs_t *aff1Inputs, affineInputs_t *aff2Inputs);
void printMatrix(float *matrix, int width, int height);
float randomRange(float min, float max);
void transferMinibatch(int minibatchSize, int batchNumber, vector<unsigned int> *indices,
                       vector<vector<float> > *x, vector<uint8_t> *y, float *dev_x,
                       unsigned int *dev_y);
using namespace std;

int main(int argc, char *argv[]) {
    char filename[256];
    char *endptr;
    if (argc < 2) {
        fprintf(stderr, "Too few command line arguments. <pathToDataset> is required\n");
        return EXIT_FAILURE;
    }
    strcpy(filename, argv[1]);

    data_t *dataset = importDataset(filename);
    printf("Dataset y size: %d\n", dataset->yTest.size());
    // Should have a training dataset and a validation dataset. x are our inputs, y are the
    // expected outputs for each given input

    // Normalization for training speed
    vector<float> means(dataset->xTrain[0].size(), 0.0);
    for (int image = 0; image < dataset->xTrain.size(); image++) {
        for (int pixel = 0; pixel < means.size(); pixel++) {
            means[pixel] += dataset->xTrain[image][pixel];
        }
    }
    // Calculate the mean
    for (int pixel = 0; pixel < means.size(); pixel++) {
        means[pixel] = means[pixel] / dataset->xTrain.size();
    }
    // Normalize to 0 mean
    for (int image = 0; image < dataset->xTrain.size(); image++) {
        for (int pixel = 0; pixel < means.size(); pixel++) {
            dataset->xTrain[image][pixel] -= means[pixel];
        }
    }
    for (int image = 0; image < dataset->xVal.size(); image++) {
        for (int pixel = 0; pixel < means.size(); pixel++) {
            dataset->xVal[image][pixel] -= means[pixel];
        }
    }
    for (int image = 0; image < dataset->xTest.size(); image++) {
        for (int pixel = 0; pixel < means.size(); pixel++) {
            dataset->xTest[image][pixel] -= means[pixel];
        }
    }

    // ********* Construct the network *************
    // The network is essentially constructed from memory allocated to store all the data as it
    // propagates through the different layers, and the kernels that implement the different
    // layers, affine, ReLu, softmax, convolutional

    // Allocate memory for all intermediate steps on the GPU. This includes caching inputs to
    // each layer, outputs, and gradients used for backpropagation Input layer

    // Input layer
    float *dev_x;
    gpuErrchk(hipMalloc((float **)&dev_x, sizeof(float) * MINIBATCHSIZE * INPUTSIZE));

    // First affine layer, creates HIDDENLAYERSIZE outputs
    affineInputs_t *aff1Inputs = affineInit(HIDDENLAYERSIZE, MINIBATCHSIZE, INPUTSIZE, dev_x);

    // ReLU intermediate layer
    reluInput_t *reluInputs = reluInit(aff1Inputs->f, HIDDENLAYERSIZE * MINIBATCHSIZE);

    // Second affine layer
    affineInputs_t *aff2Inputs =
        affineInit(CLASSES, MINIBATCHSIZE, HIDDENLAYERSIZE, reluInputs->outputs);

    // Softmax loss layer
    softmaxLoss_t *softmaxInputs = softmaxInit(CLASSES, MINIBATCHSIZE, aff2Inputs->f);

    // Hyperparameter tuning section
    int paramaterFindingIterations = 1;
    for (int iteration = 0; iteration < paramaterFindingIterations; iteration++) {
        float reg = 9e-1;        // pow(10.0, randomRange(-1, 0));
        float learnRate = 1e-4;  // pow(10.0, randomRange(-5, -3));
        printf("reg: %.1e, learn: %.1e\n", reg, learnRate);

        // ****** Initialize Model Parameters *********

        // W1 needs to be set to small values, gaussian distribution, 0 mean
        float weightScale = 0.001;
        int W1Size = sizeof(float) * HIDDENLAYERSIZE * INPUTSIZE;
        float *host_W1 = (float *)malloc(W1Size);
        std::default_random_engine generator;
        std::normal_distribution<float> distribution(0.0, weightScale);

        for (int i = 0; i < HIDDENLAYERSIZE * INPUTSIZE; i++) {
            host_W1[i] = distribution(generator);
        }
        // Copy W1 to device
        gpuErrchk(hipMemcpy(aff1Inputs->W, host_W1, W1Size, hipMemcpyHostToDevice));
        free(host_W1);

        // b1 needs to be set to 0 for no offsets at first
        int b1Size = sizeof(float) * HIDDENLAYERSIZE;
        float *host_b1 = (float *)malloc(b1Size);
        for (int i = 0; i < HIDDENLAYERSIZE; i++) {
            host_b1[i] = 0.0;
        }
        // Copy b1 to device
        gpuErrchk(hipMemcpy(aff1Inputs->b, host_b1, b1Size, hipMemcpyHostToDevice));
        free(host_b1);

        // W2 needs to be set to small values, gaussian distribution, 0 mean
        int W2Size = sizeof(float) * CLASSES * HIDDENLAYERSIZE;
        float *host_W2 = (float *)malloc(W2Size);

        for (int i = 0; i < CLASSES * HIDDENLAYERSIZE; i++) {
            host_W2[i] = distribution(generator);
        }
        // Copy W2 to device
        gpuErrchk(hipMemcpy(aff2Inputs->W, host_W2, W2Size, hipMemcpyHostToDevice));
        free(host_W2);

        // b1 needs to be set to 0 for no offsets at first
        int b2Size = sizeof(float) * CLASSES;
        float *host_b2 = (float *)malloc(b2Size);
        for (int i = 0; i < CLASSES; i++) {
            host_b2[i] = 0.0;
        }
        // Copy b2 to device
        gpuErrchk(hipMemcpy(aff2Inputs->b, host_b2, b2Size, hipMemcpyHostToDevice));
        free(host_b2);
        // ******** Start of Optimization ************

        learnParams_t learnParameters;
        learnParameters.learningRate = learnRate;
        learnParameters.momentumDecay = MOMENTUMDECAY;
        learnParameters.regStrength = reg;

        double ttrainstart = omp_get_wtime();

        // keep track of best params
        // float bAcc, bLoss,

        // Train for this many epochs
        for (int epoch = 0; epoch < NUMEPOCHS; epoch++) {
            float runningAccuracy = 0.0;
            float runningLoss = 0.0;
            float runningRegLoss = 0.0;

            // Generate series of random numbers to fill minibatches from. Only generate it once
            // per epoch
            std::vector<unsigned int> indices(dataset->yTrain.size());
            std::iota(indices.begin(), indices.end(), 0);
            std::random_shuffle(indices.begin(), indices.end());

            // Iterate through as many minibatches as we need to complete an entire epoch
            int numBatches = ceil(1.0 * dataset->yTrain.size() / MINIBATCHSIZE);
            for (int batch = 0; batch < numBatches; batch++) {
                printf("Epoch: %d, Minibatch (%d/%d)\n", epoch, batch + 1, numBatches);

                //  Sample a minibatch of samples from training data
                transferMinibatch(MINIBATCHSIZE, batch, &indices, &dataset->xTrain,
                                  &dataset->yTrain, dev_x, softmaxInputs->y);

                // Run forward and backward passes on minibatch of data, and update the gradient
                forward(aff1Inputs, reluInputs, aff2Inputs);

                // This layer computes the loss and the gradient of the loss with respect to the
                // scores input to this layer
                softmaxLoss(softmaxInputs);

                // At this point we will have the loss computed for every input image, and the
                // gradient of our softmax function. We now begin to backpropogate the gradients

                // Evaluate gradient for affine layer with respect to W and b f(x)=W*x+b, given
                // the upstream gradients and the last inputs
                backward(learnParameters.regStrength, softmaxInputs->dLdf, aff1Inputs, reluInputs,
                         aff2Inputs);

                // Using our learning rate, update our parameters based on the gradient
                update(&learnParameters, aff1Inputs, aff2Inputs);

                // gpuErrchk(hipMemcpy(host_b1, aff1Inputs->b, b1Size,
                // hipMemcpyDeviceToHost));
                //  printf("b\n");
                //  for (int i = 0; i < CLASSES; i++) {
                //      printf("%f, ", host_b1[i]);
                //  }
                //  printf("\nExpected Classes were:\n");
                //  for (int i = 0; i < MINIBATCHSIZE; i++) {
                //     printf("%d, ", minibatchY[i]);
                // }
                //  Copy f
                //  float *host_f = (float *)malloc(sizeof(float) * MINIBATCHSIZE * CLASSES);
                //  gpuErrchk(hipMemcpy(host_f, aff1Inputs->f, sizeof(float) * MINIBATCHSIZE *
                //  CLASSES,
                //                      hipMemcpyDeviceToHost));
                //  printf("\nf\n");
                //  printMatrix(host_f, MINIBATCHSIZE, CLASSES);

                // Pull accuracy
                float softmaxLoss;
                gpuErrchk(hipMemcpy(&softmaxLoss, softmaxInputs->loss, sizeof(float),
                                     hipMemcpyDeviceToHost));
                float regLoss;
                gpuErrchk(hipMemcpy(&regLoss, aff1Inputs->regLoss, sizeof(float),
                                     hipMemcpyDeviceToHost));
                float accuracy;
                gpuErrchk(hipMemcpy(&accuracy, softmaxInputs->accuracy, sizeof(float),
                                     hipMemcpyDeviceToHost));
                // printf("Batch Accuracy: %.2f\n", accuracy);
                runningAccuracy += accuracy;
                runningLoss += softmaxLoss + regLoss;
                runningRegLoss += regLoss;
            }
            runningAccuracy = runningAccuracy / numBatches;

            printf("Averaged Accuracy: %f\n", runningAccuracy);
            // printf("Averaged Loss: %f\n", runningLoss);
            // printf("Regularization Loss: %f\n", runningRegLoss);
        }
        double ttrainend = omp_get_wtime();
        printf("Training Time: %f\n", ttrainend - ttrainstart);

        // Cross validate on validation dataset

        float validationAccuracy = 0;
        // Can use this to shuffle minibatches, but since we are just validating, no reason to
        std::vector<unsigned int> indices(dataset->yVal.size());
        std::iota(indices.begin(), indices.end(), 0);

        // Iterate through as many minibatches as we need to complete an entire epoch
        int numBatches = ceil(1.0 * dataset->yVal.size() / MINIBATCHSIZE);
        for (int batch = 0; batch < numBatches; batch++) {
            // printf("Minibatch (%d/%d)\n", batch+1, numBatches);

            //  Sample a minibatch of samples from training data
            transferMinibatch(MINIBATCHSIZE, batch, &indices, &dataset->xVal, &dataset->yVal, dev_x,
                              softmaxInputs->y);

            // Run forward and backward passes on minibatch of data, and update the gradient
            forward(aff1Inputs, reluInputs, aff2Inputs);

            // This layer computes the loss and the gradient of the loss with respect to the
            // scores input to this layer
            softmaxLoss(softmaxInputs);

            // Read accuracy off
            float valAccuracy;
            gpuErrchk(hipMemcpy(&valAccuracy, softmaxInputs->accuracy, sizeof(float),
                                 hipMemcpyDeviceToHost));
            // printf("Val Batch Accuracy: %.4f\n", valAccuracy);
            validationAccuracy += valAccuracy;
        }
        printf("Averaged Val Accuracy: %f\n", validationAccuracy / numBatches);
    }
    // Evaluate test accuracy at end if requested. Don't want to peek and optimize around this
    if (TEST == 1) {
        float TestAccuracy = 0;
        // Can use this to shuffle minibatches, but since we are just validating, no reason to
        std::vector<unsigned int> indices(dataset->yTest.size());
        std::iota(indices.begin(), indices.end(), 0);

        // Iterate through as many minibatches as we need to complete an entire epoch
        int numBatches = ceil(1.0 * dataset->yTest.size() / MINIBATCHSIZE);
        for (int batch = 0; batch < numBatches; batch++) {
            // printf("Minibatch (%d/%d)\n", batch+1, numBatches);

            //  Sample a minibatch of samples from training data
            transferMinibatch(MINIBATCHSIZE, batch, &indices, &dataset->xTest, &dataset->yTest,
                              dev_x, softmaxInputs->y);

            // Run forward and backward passes on minibatch of data, and update the gradient
            forward(aff1Inputs, reluInputs, aff2Inputs);

            // This layer computes the loss and the gradient of the loss with respect to the
            // scores input to this layer
            softmaxLoss(softmaxInputs);

            // Read accuracy off
            float testAcc;
            gpuErrchk(hipMemcpy(&testAcc, softmaxInputs->accuracy, sizeof(float),
                                 hipMemcpyDeviceToHost));
            // printf("Test Batch Accuracy: %.4f\n", valAccuracy);
            TestAccuracy += testAcc;
        }
        printf("Averaged Test Accuracy: %f\n", TestAccuracy / numBatches);
    }
    // Cleanup, free memory etc
}

/*! \brief Compute the forward pass
 *
 *  Used during training as well as for evaluating model performance. Evaluate forward pass for
 * entire network
 *
 * \param aff1Inputs Inputs for first affine layer
 * \return void
 */
void forward(affineInputs_t *aff1Inputs, reluInput_t *reluInputs, affineInputs_t *aff2Inputs) {
    // Compute f(x)=W1*x+b1 forward pass
    affineForward(aff1Inputs);
    reluForward(reluInputs);
    affineForward(aff2Inputs);
}

void backward(float regStrength, float *dLdf, affineInputs_t *aff1Inputs, reluInput_t *reluInputs,
              affineInputs_t *aff2Inputs) {
    affineBackward(regStrength, dLdf, aff2Inputs);
    reluBackward(aff2Inputs->dLdx, reluInputs);
    affineBackward(regStrength, reluInputs->dLdin, aff1Inputs);
}

void update(learnParams_t *learnParameters, affineInputs_t *aff1Inputs,
            affineInputs_t *aff2Inputs) {
    affineUpdate(learnParameters, aff1Inputs);
    affineUpdate(learnParameters, aff2Inputs);
}

void printMatrix(float *matrix, int width, int height) {
    int i, j;
    int cnt = 0;
    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            printf("%+.2f, ", matrix[cnt]);
            cnt++;
        }
        printf("\n");
    }
}

float randomRange(float min, float max) {
    int range = max - min;
    // Random number [0-1)
    float num = rand() / (RAND_MAX + 1.);
    num = num * range + min;
    // printf("Random Number is: %.2f\n", num);
    return num;
}

// Copies a minibatch of data over from x and y into pre allocated buffers for x and y to be
// later copied to device. Pass in the indices of the larger set you wish to use. The length of
// indices needs to match the minibatch size. It will need to be padded at the end of the
// minibatch as well.
void transferMinibatch(int minibatchSize, int batchNumber, vector<unsigned int> *indices,
                       vector<vector<float> > *x, vector<uint8_t> *y, float *dev_x,
                       unsigned int *dev_y) {
    // Allocate host memory space to create minibatch in
    unsigned int minibatchXSize = sizeof(float) * minibatchSize * INPUTSIZE;
    float *minibatchX = (float *)malloc(minibatchXSize);
    unsigned int minibatchYSize = sizeof(unsigned int) * minibatchSize;
    unsigned int *minibatchY = (unsigned int *)malloc(minibatchYSize);

    // Sample a minibatch based on the passed in indices
    for (int i = 0; i < minibatchSize; i++) {
        int indice = batchNumber * minibatchSize + i;
        // The last minibatch will have empty slots to fill with input data, wrap back
        // around for simplicity
        if (indice >= y->size()) {
            indice = indice % y->size();
        }
        int randomIndice = (*indices)[indice];
        //  Copy over the entire vector
        for (int dim = 0; dim < INPUTSIZE; dim++) {
            // Need to push it on in a transposed fashion. Can't just push it row by
            // row, because the matrix x is really transposed from that orientation.
            minibatchX[minibatchSize * dim + i] = (*x)[randomIndice][dim];
        }
        minibatchY[i] = (*y)[randomIndice];
    }
    // Push minibatch to GPU. Push images and expected classes
    gpuErrchk(hipMemcpy(dev_x, minibatchX, minibatchXSize, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_y, minibatchY, minibatchYSize, hipMemcpyHostToDevice));

    free(minibatchX);
    free(minibatchY);
}
