#include "hip/hip_runtime.h"
// To run this program, pass in the file path to the dataset and the neural network will train on
// the dataset, and then the accuracy will be evaluated.

#include <complex.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h> /* time */
#include <unistd.h>

#include <iostream>
#include <random>

#include "affineLayer.h"
#include "softmaxLoss.h"

// Run everything on CPU
// If not defined, will run GPU implementation
#define CPU

// Number of classes to predict for on output layer
#define CLASSES 10
// Size of the NN input layer (The size of the flattened image)
#define INPUTSIZE 3072
#define TRAINSIZE 10000
#define MINIBATCHSIZE 1000
#define NUMEPOCHS 100

// Hyper parameters
#define LEARNINGRATE 0.001
#define ALPHA 0.00001
#define MOMENTUMDECAY 0.75

/*! \struct _learnParams_t
 *  \brief Hyper parameters for gradient descent
 *
 *  Contains all the hyper parameters for performing gradient descent. Uses a momentum based
 * approach with exponential decay of old gradient
 */
typedef struct _learnParams_t {
    float learningRate;  /*!< Gradient step size */
    float momentumDecay; /*!< Gradient decay for momentum */
    float regStrength;   /*!< Regularization strength for fully connected layers */
} learnParams_t;

// Error checking GPU calls
#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

using namespace std;

int main(int argc, char *argv[]) {
    // Do some argument parsing
    // Parse out the filename

    // Read in dataset from file
    // Should have a training dataset, a validation dataset, and a test dataset

    // ********* Construct the network *************
    // The network is essentially constructed from memory allocated to store all the data as it
    // propagates through the different layers, and the kernels that implement the different layers,
    // affine, ReLu, softmax, convolutional

    // Allocate memory for all intermediate steps on the GPU. This includes caching inputs to each
    // layer, outputs, and gradients used for backpropagation Input layer
    float *dev_x1;
    gpuErrchk(hipMalloc((float **)&dev_x1, sizeof(float) * MINIBATCHSIZE * INPUTSIZE));

    // W1. The weight matrix we are trying to find
    float *dev_W1;
    gpuErrchk(hipMalloc((float **)&dev_W1, sizeof(float) * CLASSES * INPUTSIZE));

    // b1. The biases for each output of the linear classifier. The +b term
    float *dev_b1;
    gpuErrchk(hipMalloc((float **)&dev_b1, sizeof(float) * CLASSES));

    // Intermediate Scores f(x). The linear classifier's predicted scores f(x)=W*x+b
    float *dev_f1;
    gpuErrchk(hipMalloc((float **)&dev_f1, sizeof(float) * CLASSES));

    AffineInputs *aff1Inputs;
    aff1Inputs->W = dev_W1;
    aff1Inputs->x = dev_x1;
    aff1Inputs->b = dev_b1;
    aff1Inputs->f = dev_f1;
    aff1Inputs->batchSize = MINIBATCHSIZE;
    aff1Inputs->dataSize = INPUTSIZE;
    aff1Inputs->numOutputs = CLASSES;

    // dL/dW1. How much the weights effect the loss
    float *dev_dLdW1;
    gpuErrchk(hipMalloc((float **)&dev_dLdW1, sizeof(float) * CLASSES * INPUTSIZE));

    // dL/db1. How much the biases effect the loss
    float *dev_dLdb1;
    gpuErrchk(hipMalloc((float **)&dev_dLdb1, sizeof(float) * CLASSES));

    AffineGradients *aff1Grads;
    aff1Grads->dLdB = dev_dLdb1;
    aff1Grads->dLdW = dev_dLdW1;

    // The expected classes of the minibatch, used to train the model
    float *dev_y;
    gpuErrchk(hipMalloc((float **)&dev_y, sizeof(float) * MINIBATCHSIZE));

    // Softmax loss
    float *dev_softmax_loss;
    gpuErrchk(hipMalloc((float **)&dev_softmax_loss, sizeof(float)));

    // Softmax dL/df. How much the loss changes with respect to each class score from the last layer
    float *dev_dLdf;
    gpuErrchk(hipMalloc((float **)&dev_dLdf, sizeof(float) * CLASSES));

    softmaxLoss_t *softmaxInputs;
    softmaxInputs->loss = dev_softmax_loss;
    softmaxInputs->dLdf = dev_dLdf;
    softmaxInputs->f = dev_f1;
    softmaxInputs->numClasses = CLASSES;
    softmaxInputs->batchSize = MINIBATCHSIZE;

    // ****** Initialize Model Parameters *********

    // W1 needs to be set to small values, gaussian distribution, 0 mean
    float weightScale = 0.001;
    int W1Size = sizeof(float) * CLASSES * INPUTSIZE;
    float *host_W1 = (float *)malloc(W1Size);
    std::default_random_engine generator;
    std::normal_distribution<float> distribution(0.0, weightScale);

    for (int i = 0; i < CLASSES * INPUTSIZE; i++) {
        host_W1[i] = distribution(generator);
    }
    // Copy W1 to device
    gpuErrchk(hipMemcpy(dev_W1, host_W1, W1Size, hipMemcpyHostToDevice));

    // b1 needs to be set to 0 for no offsets at first
    int b1Size = sizeof(float) * CLASSES;
    float *host_b1 = (float *)malloc(b1Size);
    for (int i = 0; i < CLASSES; i++) {
        host_b1[i] = 0.0;
    }
    // Copy b1 to device
    gpuErrchk(hipMemcpy(dev_b1, host_b1, b1Size, hipMemcpyHostToDevice));

    // ******** Start of Optimization ************

    learnParams_t *learnParameters;
    learnParameters->learningRate = LEARNINGRATE;
    learnParameters->momentumDecay = MOMENTUMDECAY;
    learnParameters->regStrength = ALPHA;

    // Descend gradient for this many epochs
    for (int epoch = 0; epoch < NUMEPOCHS; epoch++) {
        // Iterate through as many minibatches as we need to complete an entire epoch
        for (int batch = 0; batch < ceil(1.0 * TRAINSIZE / MINIBATCHSIZE); batch++) {
            // Sample a minibatch of samples from training data

            // Push minibatch to GPU. Push images and expected classes
            // gpuErrchk(hipMemcpy(dev_inputLayer, &trainData[minibatchStartIndex], sizeof(float) *
            // MINIBATCHSIZE);

            // Run forward and backward passes on minibatch of data, and update the gradient

            // Compute f(x)=W1*x+b1 forward pass
            dim3 blockDim(32, 32);
            // Number of threads is the size of the output matrix
            dim3 gridDim(ceil(1.0 * MINIBATCHSIZE / blockDim.x), ceil(1.0 * CLASSES / blockDim.y));
            affineForward<<<gridDim, blockDim>>>(affine1Inputs);

            // This layer computes the loss and the gradient of the loss with respect to the scores
            // input to this layer
            dim3 blockDim(32, 32);
            // Number of threads is the size of the output matrix of scores
            dim3 gridDim(ceil(1.0 * MINIBATCHSIZE / blockDim.x), ceil(1.0 * CLASSES / blockDim.y));
            softmaxLoss<<<gridDim, blockDim>>>(softmaxInputs);

            // At this point we will have the loss computed for every input image, and the gradient
            // of our softmax function. We now begin to backpropogate the gradients

            // Evaluate gradient for affine layer with respect to W and b f(x)=W*x+b, given the
            // upstream gradients and the last inputs
            dim3 blockDim(32, 32);
            // Number of threads is the size of the output matrix
            dim3 gridDim(ceil(1.0 * MINIBATCHSIZE / blockDim.x), ceil(1.0 * CLASSES / blockDim.y));
            affineBackward<<<gridDim, blockDim>>>(dev_dLdf, aff1Inputs, aff1Grads);

            // Using our learning rate, update our parameters based on the gradient

            // Update Affine1 layer weights
            dim3 blockDim(32, 32);
            dim3 gridDim(ceil(1.0 * MINIBATCHSIZE / blockDim.x), ceil(1.0 * CLASSES / blockDim.y));
            affineUpdate<<<gridDim, blockDim>>>(learnParameters, aff1Inputs, aff1Grads);

            // Print out the loss for debugging
            float loss;
            gpuErrchk(hipMemcpy(&loss, dev_softmax_loss, sizeof(float), hipMemcpyDeviceToHost));
            printf("\nSoftmax Loss: %f", loss);
        }
    }

    // TODO Optional, save model off so we don't have to retrain in the future

    // Evaluate accuracy of classifier on training dataset

    // Evaluate accuracy of classifier on validation dataset

    // Evaluate accuracy of classifier on test dataset (Don't really need to do this since this is
    // mostly just for fun)

    // Cleanup, free memory etc
}
